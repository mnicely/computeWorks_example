#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2019 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

/*
 * This sample compares performance between serial matrix multiplication and
 * an optimized CUDA kernel.
 */

/*
 * nvcc -O2 cuda.cu -o cuda -run
 */

#include <chrono>
#include <cmath>
#include <hip/hip_cooperative_groups.h>
#include <cstdio>
#include <cstdlib>

// Thread block size
#define BLOCK_SIZE 16

// Verify input matrices
void verify( int const &n, float const *C_ref, float const *C_test ) {

    // Check result against reference
    float error_norm = 0.0f;
    float ref_norm   = 0.0f;
    float diff       = 0.0f;

    for ( int i = 0; i < n * n; i++ ) {
        diff = C_test[i] - C_ref[i];
        error_norm += diff * diff;
        ref_norm += C_test[i] * C_test[i];
    } // i

    error_norm = static_cast<float>( std::sqrt( static_cast<double>( error_norm ) ) );
    ref_norm   = static_cast<float>( std::sqrt( static_cast<double>( ref_norm ) ) );

    if ( std::fabs( ref_norm ) < 1e-7 )
        std::printf( "Reference norm is 0.\t" );

    if ( error_norm / ref_norm < 1e-5f )
        std::printf( "Test passed.\n" );
    else
        std::printf( "Test failed.\n" );
} // verify

void normalC( int const &  n,
              float const &alpha,
              float const *A,
              float const *B,
              float const &beta,
              float *      C,
              int const &  loops ) {

    auto start = std::chrono::high_resolution_clock::now( );

    for ( int l = 0; l < loops; l++ ) {
        for ( int i = 0; i < n; ++i ) {
            for ( int j = 0; j < n; ++j ) {
                float prod = 0.0f;
                for ( int k = 0; k < n; ++k ) {
                    prod += A[k * n + i] * B[j * n + k];
                } // k
                C[j * n + i] = alpha * prod + beta * C[j * n + i];
            } // j
        }     // i
    }         // loops

    auto                                      stop       = std::chrono::high_resolution_clock::now( );
    std::chrono::duration<double, std::milli> elapsed_ms = stop - start;
    std::printf( "%0.2f ms\n", elapsed_ms.count( ) / loops );
} // normalC

// Matrix multiplication kernel called by MatMul()
__global__ void
MatMulKernel( int const n, float const *__restrict__ A, float const *__restrict__ B, float *__restrict__ C ) {

    auto const block = cooperative_groups::this_thread_block( );

    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread computes one element of Csub by accumulating results into Cvalue
    float Cvalue = 0.0f;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    float *Csub = &C[n * BLOCK_SIZE * blockRow + BLOCK_SIZE * blockCol];

    // Loop over all the sub-matrices of A and B that are required to compute Csub
    // Multiply each pair of sub-matrices together and accumulate the results
    for ( int m = 0; m < ( n / BLOCK_SIZE ); ++m ) {

        // Get sub-matrix Asub of A
        float const *Asub = &A[n * BLOCK_SIZE * blockRow + BLOCK_SIZE * m];

        // Get sub-matrix Bsub of B
        float const *Bsub = &B[n * BLOCK_SIZE * m + BLOCK_SIZE * blockCol];

        // Shared memory used to store Asub and Bsub respectively
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = Asub[row * n + col];
        Bs[row][col] = Bsub[row * n + col];

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        block.sync( );

        // Multiply Asub and Bsub together
        for ( int e = 0; e < BLOCK_SIZE; ++e )
            Cvalue += As[row][e] * Bs[e][col];

        // Synchronize to make sure that the preceding computation is done
        // before loading two new sub-matrices of A and B in the next iteration
        block.sync( );
    } // m

    // Write Csub to device memory each thread writes one element
    Csub[row * n + col] = Cvalue;
} // MatMulKernel

void cuda( int const &  n,
           float const &alpha,
           float const *A,
           float const *B,
           float const &beta,
           float *      C,
           int const &  loops ) {

    // Declare timer variables
    hipEvent_t startEvent = nullptr;
    hipEvent_t stopEvent  = nullptr;
    hipEventCreate( &startEvent, hipEventBlockingSync );
    hipEventCreate( &stopEvent, hipEventBlockingSync );

    // Declare device result pointers
    float *d_A, *d_B, *d_C;

    // Allocate memory on device
    hipMalloc( ( void ** )&d_A, sizeof( float ) * n * n );
    hipMalloc( ( void ** )&d_B, sizeof( float ) * n * n );
    hipMalloc( ( void ** )&d_C, sizeof( float ) * n * n );

    // Copy host memory to device
    hipMemcpy( d_A, A, sizeof( float ) * n * n, hipMemcpyHostToDevice );
    hipMemcpy( d_B, B, sizeof( float ) * n * n, hipMemcpyHostToDevice );

    // setup the dimensions
    dim3 blocksPerGrid( ( n + BLOCK_SIZE - 1 ) / BLOCK_SIZE, ( n + BLOCK_SIZE - 1 ) / BLOCK_SIZE );
    dim3 threadsPerBlock( BLOCK_SIZE, BLOCK_SIZE );

    hipEventRecord( startEvent );

    for ( int l = 0; l < loops; l++ )
        MatMulKernel<<<blocksPerGrid, threadsPerBlock>>>( n, d_A, d_B, d_C );

    hipDeviceSynchronize( );
    hipEventSynchronize( stopEvent );

    // Copy results from device to host
    hipMemcpy( C, d_C, sizeof( float ) * n * n, hipMemcpyDeviceToHost );

    hipFree( d_A );
    hipFree( d_B );
    hipFree( d_C );

    float elapsed_ms;
    hipEventElapsedTime( &elapsed_ms, startEvent, stopEvent );
    std::printf( "%0.2f ms\n", elapsed_ms / loops );
} // cuda

int main( int argc, char **argv ) {

    int n = 1024;
    if ( argc > 1 )
        n = std::atoi( argv[1] );
    printf( "Running with N = %d\n\n", n );

    float alpha = 1.0f;
    float beta  = 0.0f;

    // Declare host variables
    float *h_A      = new float[sizeof( float ) * n * n];
    float *h_B      = new float[sizeof( float ) * n * n];
    float *h_C      = new float[sizeof( float ) * n * n];
    float *h_C_cuda = new float[sizeof( float ) * n * n];

    // Initialize values
    for ( int i = 0; i < n * n; i++ ) {
        h_A[i] = 2.0f;
        h_B[i] = 1.0f;
    } // i

    // Benchmark normal C matrix multiplication
    printf( "Running Normal C: " );
    normalC( n, alpha, h_A, h_B, beta, h_C, 2 );

    // Benchmark and verify CUDA matrix multiplication
    printf( "Running CUDA: " );
    cuda( n, alpha, h_A, h_B, beta, h_C_cuda, 5 );
    verify( n, h_C, h_C_cuda );

    // Memory clean up
    delete[]( h_A );
    delete[]( h_B );
    delete[]( h_C );
    delete[]( h_C_cuda );
} // main
